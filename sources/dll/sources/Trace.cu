#include "hip/hip_runtime.h"
#include "Trace.cuh"

#include "hip/hip_runtime.h"
#include ""

#include "Camera.h"
#include "LightSource.h"
#include "Triangle.h"

Camera **dev_cameras;
Triangle **dev_triangles;
LightSource **dev_lights;

hipError_t CopyToDevice(Scene * s)
{
	hipError_t e = hipSuccess;
	dev_triangles = new Triangle*[s->triangles.size()];
	for (int i = 0; i < s->triangles.size(); i++)
	{
		e = hipMalloc((void**)&dev_triangles[i], sizeof(Triangle));
		if (e != hipSuccess)
		{
			return e;
		}
		e = hipMemcpy(dev_triangles[i], s->triangles[i], sizeof(Triangle), hipMemcpyHostToDevice);
		if (e != hipSuccess)
		{
			return e;
		}
	}

	return e;
}

void startSequential()
{
	SequentialTrace << <1, 1 >> > (dev_triangles);
}

__global__ void SequentialTrace(Triangle **dev_triangles)
{
	printf("%d", dev_triangles[0]->p0);
}

__global__ void ParallelTrace()
{

}

hipError_t CopyFromDevice(Scene * s)
{
	hipError_t e = hipSuccess;

	return e;
}