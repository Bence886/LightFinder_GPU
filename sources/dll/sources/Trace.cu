#include "hip/hip_runtime.h"
#include "Trace.cuh"

#include "hip/hip_runtime.h"
#include ""

#include "Camera.h"
#include "LightSource.h"
#include "Triangle.h"
#include "Point.h"

Camera *dev_cameras;
LightSource *dev_lights;
Triangle *dev_triangles;

hipError_t CopyToDevice(Scene * s)
{
	hipError_t e = hipSuccess;
	e = hipMalloc((void**)&dev_triangles, sizeof(Triangle) * s->triangles.size());
	if (e != hipSuccess)
	{
		return e;
	}
	for (int i = 0; i < s->triangles.size(); i++)
	{
		e = hipMemcpy(&(dev_triangles[i]), s->triangles[i], sizeof(Triangle), hipMemcpyHostToDevice);
		if (e != hipSuccess)
		{
			return e;
		}
	}

	e = hipMalloc((void**)&dev_lights, sizeof(LightSource) * s->lights.size());
	if (e != hipSuccess)
	{
		return e;
	}
	for (int i = 0; i < s->lights.size(); i++)
	{
		e = hipMemcpy(&(dev_lights[i]), s->lights[i], sizeof(LightSource), hipMemcpyHostToDevice);
		if (e != hipSuccess)
		{
			return e;
		}
	}

	e = hipMalloc((void**)&dev_cameras, sizeof(Camera) * s->cameras.size());
	if (e != hipSuccess)
	{
		return e;
	}
	for (size_t i = 0; i < s->cameras.size(); i++)
	{
		e = hipMemcpy(&(dev_cameras[0]), s->cameras[i], sizeof(Camera), hipMemcpyHostToDevice);
		if (e != hipSuccess)
		{
			return e;
		}
	}
	return e;
}

void StartSequential()
{
	SequentialTrace << <1, 1 >> > (dev_triangles, dev_lights, dev_cameras);
}

void startParallel()
{
}

__global__ void SequentialTrace(Triangle *dev_triangles, LightSource *dev_lights, Camera *dev_cameras)
{
	dev_cameras->lookDirections[0] = Point(6, 87, 79);
	printf("%f\n", dev_cameras->lookDirections[0].X);
}

__global__ void ParallelTrace()
{

}

hipError_t CopyFromDevice(Scene * s)
{
	hipError_t e = hipSuccess;
	for (int i = 0; i < s->cameras.size(); i++)
	{
		e = hipMemcpy(s->cameras[i], &(dev_cameras[i]), sizeof(Camera), hipMemcpyDeviceToHost);
		if (e != hipSuccess)
		{
			return e;
		}
	}

	return e;
}