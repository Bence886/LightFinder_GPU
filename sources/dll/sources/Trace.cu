#include "hip/hip_runtime.h"
#include "Trace.cuh"

#include "hip/hip_runtime.h"
#include ""

	#include "Camera.h"
	#include "LightSource.h"
	#include "Triangle.h"
	#include "Point.h"

Camera *dev_cameras;
LightSource *dev_lights;
Triangle *dev_triangles;

hipError_t CopyToDevice(Scene * s)
{
	hipError_t e = hipSuccess;
	e = hipMalloc((void**)&dev_triangles, sizeof(Triangle) * s->triangles.size());
	if (e != hipSuccess)
	{
		return e;
	}
	for (int i = 0; i < s->triangles.size(); i++)
	{
		e = hipMemcpy(&(dev_triangles[i]), s->triangles[i], sizeof(Triangle), hipMemcpyHostToDevice);
		if (e != hipSuccess)
		{
			return e;
		}
	}

	e = hipMalloc((void**)&dev_lights, sizeof(LightSource) * s->lights.size());
	if (e != hipSuccess)
	{
		return e;
	}
	for (int i = 0; i < s->lights.size(); i++)
	{
		e = hipMemcpy(&(dev_lights[i]), s->lights[i], sizeof(LightSource), hipMemcpyHostToDevice);
		if (e != hipSuccess)
		{
			return e;
		}
	}

	e = hipMalloc((void**)&dev_cameras, sizeof(Camera) * s->cameras.size());
	if (e != hipSuccess)
	{
		return e;
	}
	for (size_t i = 0; i < s->cameras.size(); i++)
	{
		e = hipMemcpy(&(dev_cameras[0]), s->cameras[i], sizeof(Camera), hipMemcpyHostToDevice);
		if (e != hipSuccess)
		{
			return e;
		}
	}
	return e;
}

void StartSequential()
{
	SequentialTrace << <1, 1 >> > (dev_triangles, dev_lights, dev_cameras);
}

void startParallel(int block, int thread) //cameras / sampling
{
	ParallelTrace << <block, thread >> > (dev_triangles, dev_lights, dev_cameras);
}

__global__ void SequentialTrace(Triangle *dev_triangles, LightSource *dev_lights, Camera *dev_cameras)
{
	for (int j = 0; j < 1 ; j++)
	{
		for (int i = 0; i < SAMPLING; i++)
		{
			Point ray = Triangle::GetPointOnSphere(dev_cameras[j].origin);
			Vector vector(dev_cameras[j].origin, ray);
			float a ;//= CpuTrace(dev_lights, dev_triangles, &vector, MAX_DEPT);
			ray = vector.Direction;
			ray.MultiplyByLambda(a);
			if (a != 0)
			{
				dev_cameras[j].lookDirections[dev_cameras[j].lookNum++] = ray;
			}
		}
	}
}

__global__ void ParallelTrace(Triangle *dev_triangles, LightSource *dev_lights, Camera *dev_cameras)
{

}

hipError_t CopyFromDevice(Scene * s)
{
	hipError_t e = hipSuccess;
	for (int i = 0; i < s->cameras.size(); i++)
	{
		e = hipMemcpy(s->cameras[i], &(dev_cameras[i]), sizeof(Camera), hipMemcpyDeviceToHost);
		if (e != hipSuccess)
		{
			return e;
		}
	}

	return e;
}