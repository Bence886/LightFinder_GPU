#include "hip/hip_runtime.h"
#include "Trace.cuh"

#include "hip/hip_runtime.h"
#include ""

#include "Camera.h"
#include "LightSource.h"
#include "Triangle.h"

__device__ Camera *d_cameras;
__device__ LightSource *d_lights;
__device__ Triangle *d_triangles;

hipError_t CopyToDevice(Scene * s)
{
	hipError_t e = hipMemcpyToSymbol(HIP_SYMBOL(d_cameras), s->cameras[0], s->cameras.size());
	if (e != hipError_t::hipSuccess)
	{
		return e;
	}
	e = hipMemcpyToSymbol(HIP_SYMBOL(d_lights), s->lights[0], s->lights.size());
	if (e != hipError_t::hipSuccess)
	{
		return e;
	}
	e = hipMemcpyToSymbol(HIP_SYMBOL(d_triangles), s->triangles[0], s->triangles.size());

	return e;
}

__global__ void SequentialTrace()
{
	
	//d_cameras->lookDirections[1] = d_cameras->lookDirections[0];
}

__global__ void ParallelTrace()
{

}

hipError_t CopyFromDevice(Scene * s)
{
	hipError_t e = hipMemcpyFromSymbol(s->cameras[0], HIP_SYMBOL(d_cameras), s->triangles.size());

	return e;
}

