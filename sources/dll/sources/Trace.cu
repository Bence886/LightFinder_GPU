#include "hip/hip_runtime.h"
#include "Trace.cuh"

#include "hip/hip_runtime.h"
#include ""

#include "Camera.h"
#include "LightSource.h"
#include "Triangle.h"

Camera **cameras;
LightSource **lights;
Triangle **triangles;

hipError_t CopyToDevice(Scene * s)
{
	hipError_t e;
	cameras = new Camera*[s->cameras.size()];
	int i = 0;
	for (Camera *item : s->cameras)
	{
		e = Camera::CopyToSymbol(item, cameras[i]);
		if (e != hipSuccess)
		{
			return e;
		}
		i++;
	}
	return e;
}


__global__ void SequentialTrace()
{

	//d_cameras->lookDirections[1] = d_cameras->lookDirections[0];
}

__global__ void ParallelTrace()
{

}

hipError_t CopyFromDevice(Scene * s)
{
	return hipSuccess;
}