#include "hip/hip_runtime.h"
#include "Trace.cuh"

#include "hip/hip_runtime.h"
#include ""

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include "Log.h"

#include "Camera.h"
#include "LightSource.h"
#include "Triangle.h"
#include "Point.h"

Camera *dev_cameras;
LightSource *dev_lights;
Triangle *dev_triangles;
int dev_triangles_len, dev_lights_len, dev_cameras_len;

hipError_t CopyToDevice(Scene * s)
{
	hipError_t e = hipSuccess;
	e = hipMalloc((void**)&dev_triangles, sizeof(Triangle) * s->triangles.size());
	if (e != hipSuccess)
	{
		WriteLog("Malloc dev_triangles: ", true, Log::Exception);
		return e;
	}
	for (int i = 0; i < s->triangles.size(); i++)
	{
		e = hipMemcpy(&(dev_triangles[i]), s->triangles[i], sizeof(Triangle), hipMemcpyHostToDevice);
		if (e != hipSuccess)
		{
			WriteLog("Copy dev_triangles: ", true, Log::Exception);
			return e;
		}
	}

	e = hipMalloc((void**)&dev_lights, sizeof(LightSource) * s->lights.size());
	if (e != hipSuccess)
	{
		WriteLog("Malloc dev_lights: ", true, Log::Exception);
		return e;
	}
	for (int i = 0; i < s->lights.size(); i++)
	{
		e = hipMemcpy(&(dev_lights[i]), s->lights[i], sizeof(LightSource), hipMemcpyHostToDevice);
		if (e != hipSuccess)
		{
			WriteLog("Copy dev_triangles: ", true, Log::Exception);
			return e;
		}
	}

	e = hipMalloc((void**)&dev_cameras, sizeof(Camera) * s->cameras.size());
	if (e != hipSuccess)
	{
		WriteLog("Malloc dev_cameras: ", true, Log::Exception);
		return e;
	}
	for (size_t i = 0; i < s->cameras.size(); i++)
	{
		e = hipMemcpy(&(dev_cameras[0]), s->cameras[i], sizeof(Camera), hipMemcpyHostToDevice);
		if (e != hipSuccess)
		{
			WriteLog("Copy dev_triangles: ", true, Log::Exception);
			return e;
		}
	}
	return e;

	dev_cameras_len = s->cameras.size();
	dev_triangles_len = s->triangles.size();
	dev_lights_len = s->lights.size();
}

void StartSequential()
{
	WriteLog("Started sequential GPU trace", true, Log::Trace);
	SequentialTrace << <1, 1 >> > (dev_triangles, dev_lights, dev_cameras, dev_triangles_len, dev_lights_len, dev_cameras_len);
	WriteLog("Finished sequential GPU trace", true, Log::Trace);

}

void startParallel(int block, int thread) //cameras / sampling
{
	WriteLog("Started parallel GPU trace", true, Log::Trace);
	ParallelTrace<<<block, thread>>>(dev_triangles, dev_lights, dev_cameras, dev_triangles_len, dev_lights_len, dev_cameras_len);
	WriteLog("Finished parallel GPU trace", true, Log::Trace);

}

__global__ void SequentialTrace(Triangle *dev_triangles, LightSource *dev_lights, Camera *dev_cameras, int dev_triangles_len, int dev_lights_len, int dev_cameras_len)
{
	Triangle::InitCuRand();
	for (int j = 0; j < dev_cameras_len; j++)
	{
		for (int i = 0; i < SAMPLING; i++)
		{
			printf("LookNum: %d \n", i);

			Point ray = Triangle::GetPointOnSphere(dev_cameras[j].origin);
			Vector vector(dev_cameras[j].origin, ray);
			float a = Trace(dev_lights, dev_triangles, &vector, MAX_DEPT, dev_triangles_len, dev_lights_len, dev_cameras_len);
			ray = vector.Direction;
			ray.MultiplyByLambda(a);
			if (a != 0)
			{
				dev_cameras[j].lookDirections[dev_cameras[j].lookNum++] = ray;
				printf("%f\n", dev_cameras[j].lookDirections[i].X);
			}
		}
	}
}

__device__ float Trace(LightSource* dev_lights, Triangle *dev_triangles, Vector *startPoint, int dept, int dev_triangles_len, int dev_lights_len, int dev_cameras_len)
{
	for (int i = 0; i < dept; i++)
	{
		LightSource **directHitLights = new LightSource*[dev_lights_len];
		Point rayToPoint;
		int j = 0;
		for(int k = 0; k < dev_lights_len; k++)
		{
			rayToPoint = dev_lights[k].location - startPoint->Location;
			rayToPoint.Normalize();
			if (Camera::LightHitBeforeTriangle(dev_lights[k], dev_triangles, Vector(startPoint->Location, rayToPoint), dev_triangles_len))
			{
				directHitLights[j++] = &dev_lights[k];
			}
		}
		if (j > 0)
		{
			int max = 0;
			int idx = 0;
			for (int k = 0; k < dev_lights_len; k++)
			{
				LightSource *aktLight = directHitLights[0];
				if (directHitLights[k] && aktLight->intensity < directHitLights[k]->intensity )
				{
					max = idx;
				}
				idx++;
			}
			startPoint->Direction = rayToPoint;
			LightSource *aktLight = directHitLights[0];
			return aktLight->intensity;
		}
		std::pair<Triangle*, Point*> *trianglePointPair = Triangle::ClosestTriangleHit(dev_triangles, *startPoint, dev_triangles_len);

		if (trianglePointPair->first && trianglePointPair->second)
		{
			Triangle triangleHit = *trianglePointPair->first;
			Point pointHit = *trianglePointPair->second;
			Point offset(startPoint->Direction);
			offset.MultiplyByLambda(-1);
			offset.MultiplyByLambda(0.001f);
			pointHit = pointHit + offset;

			bool backfacing = Point::DotProduct(triangleHit.normal, startPoint->Direction) > 0;

			startPoint = &Vector(pointHit, Triangle::GetPointOnHalfSphere(triangleHit, backfacing));
		}
	}
	return 0;
}

__global__ void ParallelTrace(Triangle *dev_triangles, LightSource *dev_lights, Camera *dev_cameras, int dev_triangles_len, int dev_lights_len, int dev_cameras_len)
{

}

hipError_t CopyFromDevice(Scene * s)
{
	hipError_t e = hipSuccess;
	for (int i = 0; i < s->cameras.size(); i++)
	{
		e = hipMemcpy(s->cameras[i], &(dev_cameras[i]), sizeof(Camera), hipMemcpyDeviceToHost);
		if (e != hipSuccess)
		{
			WriteLog("Copy back dev_cameras: ", true, Log::Error);
			return e;
		}
	}

	return e;
}