#include "kernel.cuh"

#include <iostream>

#include "hip/hip_runtime.h"
#include ""

#include "Log.h"

#define OUTPUT_NAME "Blender.txt"
#define INPUT_NAME "In.xml"

Scene *myScene;

void ProcessInput()
{
	WriteLog(std::string("Started reading input from: ") + INPUT_NAME, true, Log::Debug);
	myScene = new Scene(INPUT_NAME);
	WriteLog(std::string("Finished reading input from: ") + INPUT_NAME, true, Log::Debug);
}

void StartCPU()
{
	myScene->StartTrace_CPU();
	WriteLog("Finished CPU trace.", true, Log::Debug);
}

void StartGPU()
{
}

void WriteOutput()
{
	WriteLog(std::string("Started writing belnder scripts to: ") + OUTPUT_NAME, true, Log::Debug);
	BlenderScriptCreator bs(OUTPUT_NAME);

	for (Camera *item : myScene->cameras)
	{
		bs.CreateObject(item->lookDirections, "Camera");
	}
	WriteLog(std::string("Finished writing belnder scripts to: ") + OUTPUT_NAME, true, Log::Debug);
}

void Close()
{
	delete myScene;
}