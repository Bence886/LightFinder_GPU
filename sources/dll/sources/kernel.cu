#include "hip/hip_runtime.h"
#include "kernel.cuh"

#include <iostream>

#include "hip/hip_runtime.h"
#include ""

#include "Point.h"

__device__ Point* d_P;
Point P(1, 1, 1);

__global__ void ASD()
{
	d_P->X++;
}

void Init()
{

	hipMemcpyToSymbol(HIP_SYMBOL(d_P), &P, sizeof(Point));

	ASD << <1, 1 >> > ();

	hipMemcpyFromSymbol(&P, HIP_SYMBOL(d_P), sizeof(Point));

	std::cout << P.X << std::endl;

	std::getchar();
}
