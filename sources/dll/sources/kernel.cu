#include "kernel.cuh"

#include <iostream>

#include "hip/hip_runtime.h"
#include ""

#include "Trace.cuh"

#include "Log.h"

#define OUTPUT_NAME "Blender.txt"
#define INPUT_NAME "In.xml"

#define cudaCheckError() {hipError_t e = hipGetLastError(); WriteLog(std::string("CUDA result:") + hipGetErrorString(e), true, Log::Exception);}

Scene *myScene;

void ProcessInput()
{
	WriteLog(std::string("Started reading input from: ") + INPUT_NAME, true, Log::Debug);
	myScene = new Scene(INPUT_NAME);
	WriteLog(std::string("Finished reading input from: ") + INPUT_NAME, true, Log::Debug);
}

void StartCPU()
{
	myScene->StartTrace_CPU();
	WriteLog("Finished CPU trace.", true, Log::Debug);
}

void StartGPU()
{
	WriteLog("Started copy to GPU", true, Log::Trace);
	CopyToDevice(myScene);
	cudaCheckError();

	StartSequential();
	cudaCheckError();

	WriteLog("Started copy from GPU", true, Log::Trace);
	CopyFromDevice(myScene);
	cudaCheckError();
}

void WriteOutput()
{
	WriteLog(std::string("Started writing belnder scripts to: ") + OUTPUT_NAME, true, Log::Debug);
	BlenderScriptCreator bs(OUTPUT_NAME);

	for (Triangle *item : myScene->triangles)
	{
		bs.CreateObject(new Point[3]{item->p0, item->p1, item->p2}, "Triangle", 3);
	}

	for (LightSource *item : myScene->lights)
	{
		bs.CreateObject(new Point[1]{ item->location }, "Light", 1);
	}

	for (Camera *item : myScene->cameras)
	{
		bs.CreateObject(item->lookDirections, "Camera", item->lookNum);
	}
	WriteLog(std::string("Finished writing belnder scripts to: ") + OUTPUT_NAME, true, Log::Debug);
}

void Close()
{
	delete myScene;
}